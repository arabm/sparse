#include "hip/hip_runtime.h"



#include "cs_sparse_cuda.cuh"

#include "stdio.h"

int cs_gaxpy_cuda(const cs *A, const double *x, double *y){
    int n = A->n,m = A->m, *Ai = A->i,*Ap = A->p, nzmax = A->nzmax;
    
    const double *Ax = A->x;
  
	 cs dA;
	memcpy(&dA,A,sizeof(cs));

	hipMalloc(&(dA.p), (n+1)*sizeof(int));
	hipMemcpy(dA.p,Ap,(n+1)*sizeof(int),hipMemcpyHostToDevice);

	hipMalloc(&(dA.i), nzmax*sizeof(int));
	hipMemcpy(dA.i,Ai,nzmax*sizeof(int),hipMemcpyHostToDevice);
	
	hipMalloc(&(dA.x), nzmax*sizeof(double));
	hipMemcpy(dA.x,Ax,nzmax*sizeof(double),hipMemcpyHostToDevice);
		
	double *dx;
	hipMalloc(&dx,sizeof(double)*n);
	hipMemcpy(dx,x,sizeof(double)*n,hipMemcpyHostToDevice);
	double *dy;
	hipMalloc(&dy,sizeof(double)*n);
	hipMemcpy(dy,y,sizeof(double)*n,hipMemcpyHostToDevice);

    dim3 sGrid(1,1,1);
    dim3 sBloc(m,1,1);
    
	cs_gaxpy_cuda_kernel<<<sGrid,sBloc>>>(dA,dx,dy);

	hipMemcpy(y,dy,sizeof(double)*n,hipMemcpyDeviceToHost);
	hipFree(dx);
	hipFree(dy);
	hipFree(dA.i);
	hipFree(dA.p);
	hipFree(dA.x);
	hipFree(&dA);
//	hipDeviceReset();
    return 0;
};



__global__ void cs_gaxpy_cuda_kernel(const cs A, const double *x, double *y){

	__shared__ double sy[10];
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int idy = threadIdx.y + blockIdx.y*blockDim.y;
    
    int j,n = A.n,m=A.m, *Ai = A.i,*Ap = A.p;
    const double *Ax = A.x;
   
	if (idx<10){
		sy[idx]=y[idx];
	}
	__syncthreads();

    double tmp = 0;
	int indice = 0; 
	for (j = 0; j < n;j++){
		indice = idx+Ap[j];
		if (indice < Ap[j+1]){
	            	sy[Ai[indice]] +=  Ax[indice]*x[j];
	        }
		__syncthreads();
	}
	
	if (idx <10){
		y[idx] = sy[idx];
	}
	__syncthreads();
    
    

//    cs_gaxpy_cuda_device(A,x,y); /* Call device function */

};

__device__ void cs_gaxpy_cuda_device(const cs *A, const double *x, double *y){

    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int idy = threadIdx.y + blockIdx.y*blockDim.y;
    
    int i,j,n = A->n,m=A->m, *Ai = A->i,*Ap = A->p;
    const double *Ax = A->x;
  
  printf("Hello thread %d\n", threadIdx.x); 
	double tmp = 0; 
    if ( idx < m){
	for (j = 0; j < n;j++){
	        for (i = Ap[j]; i < Ap[j+1]; i++){
			if (idy == Ai[i])
	            		tmp += Ax[i]*x[idy];
	        }
	}
	y[idy] = 1;//tmp;
    }

};
